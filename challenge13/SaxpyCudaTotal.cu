#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) y[i] = a * x[i] + y[i];
}

int main(void)
{
  // Warm-up to initialize CUDA context (not timed)
  float *d_x_warmup, *d_y_warmup;
  hipMalloc(&d_x_warmup, 256 * sizeof(float));
  hipMalloc(&d_y_warmup, 256 * sizeof(float));
  saxpy<<<1, 256>>>(256, 2.0f, d_x_warmup, d_y_warmup);
  hipDeviceSynchronize();
  hipFree(d_x_warmup);
  hipFree(d_y_warmup);

  // Start CSV file
  FILE *fp = fopen("timing.csv", "w");
  fprintf(fp, "N,TotalTime_ms,KernelTime_ms\n");

  for (int exp = 15; exp <= 25; exp++) {
    int N = 1 << exp;

    // Total time measurement start
    hipEvent_t totalStart, totalStop;
    hipEventCreate(&totalStart);
    hipEventCreate(&totalStop);
    hipEventRecord(totalStart);

    // Host allocations
    float *x = (float*)malloc(N * sizeof(float));
    float *y = (float*)malloc(N * sizeof(float));

    // Device allocations
    float *d_x, *d_y;
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));

    // Initialize host arrays
    for (int i = 0; i < N; i++) {
      x[i] = 1.0f;
      y[i] = 2.0f;
    }

    // Copy to device
    hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

    // Kernel-only time measurement
    hipEvent_t kernelStart, kernelStop;
    hipEventCreate(&kernelStart);
    hipEventCreate(&kernelStop);

    hipEventRecord(kernelStart);
    saxpy<<<(N + 255) / 256, 256>>>(N, 2.0f, d_x, d_y);
    hipEventRecord(kernelStop);

    hipEventSynchronize(kernelStop);

    float kernelTimeMs = 0;
    hipEventElapsedTime(&kernelTimeMs, kernelStart, kernelStop);

    // Copy back to host
    hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    // Check correctness
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
      maxError = fmaxf(maxError, fabsf(y[i] - 4.0f));

    // Total time measurement stop
    hipEventRecord(totalStop);
    hipEventSynchronize(totalStop);

    float totalTimeMs = 0;
    hipEventElapsedTime(&totalTimeMs, totalStart, totalStop);

    // Print and log both times
    printf("N = 2^%d, Max error: %f, Total Time: %f ms, Kernel Time: %f ms\n",
           exp, maxError, totalTimeMs, kernelTimeMs);
    fprintf(fp, "%d,%f,%f\n", N, totalTimeMs, kernelTimeMs);

    // Cleanup
    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);

    hipEventDestroy(kernelStart);
    hipEventDestroy(kernelStop);
    hipEventDestroy(totalStart);
    hipEventDestroy(totalStop);
  }

  fclose(fp);
  return 0;
}

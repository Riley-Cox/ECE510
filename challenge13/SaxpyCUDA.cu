#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) y[i] = a * x[i] + y[i];
}

int main(void)
{
  // Warm-up to initialize CUDA context (not timed)
  float *d_x_warmup, *d_y_warmup;
  hipMalloc(&d_x_warmup, 256 * sizeof(float));
  hipMalloc(&d_y_warmup, 256 * sizeof(float));
  saxpy<<<1, 256>>>(256, 2.0f, d_x_warmup, d_y_warmup);
  hipDeviceSynchronize();
  hipFree(d_x_warmup);
  hipFree(d_y_warmup);

  // Start CSV file
  FILE *fp = fopen("timing.csv", "w");
  fprintf(fp, "N,Time_ms\n");

  for (int exp = 15; exp <= 25; exp++) {
    int N = 1 << exp;

    float *x, *y, *d_x, *d_y;
    x = (float*)malloc(N * sizeof(float));
    y = (float*)malloc(N * sizeof(float));

    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));

    for (int i = 0; i < N; i++) {
      x[i] = 1.0f;
      y[i] = 2.0f;
    }

    hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start time
    hipEventRecord(start);
    saxpy<<<(N + 255) / 256, 256>>>(N, 2.0f, d_x, d_y);
    hipEventRecord(stop);

    // Wait and measure time
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    // Check for correctness
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
      maxError = fmaxf(maxError, fabsf(y[i] - 4.0f));

    printf("N = 2^%d, Max error: %f, Time: %f ms\n", exp, maxError, milliseconds);
    fprintf(fp, "%d,%f\n", N, milliseconds);

    // Cleanup
    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);
    hipEventDestroy(start);
    hipEventDestroy(stop);
  }

  fclose(fp);
  return 0;
}
